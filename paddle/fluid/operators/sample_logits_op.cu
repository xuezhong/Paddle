#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#pragma once

#include <vector>
#include "paddle/fluid/framework/eigen.h"
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/framework/tensor_util.h"
#include "paddle/fluid/operators/math/math_function.h"
#include "paddle/fluid/operators/math/sample_prob.h"
#include "paddle/fluid/operators/math/softmax.h"
#include "paddle/fluid/operators/sample_logits_op.h"

namespace paddle {
namespace operators {

// UNDERSTAND: something like take_along_axis in numpy.
template <typename T>
__global__ void GPUTakeAlongD1(const int size,
                           const int batch_size,
                           const int num_take,
                           const int array_slice_size,
                           const int idx_slice_size,
                           const T* p_array,
                           const int64_t* p_index,
                           T* p_value) {
  const auto value_slice_size = idx_slice_size;
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  int step_size = blockDim.x * gridDim.x;

  for (; idx < size; idx += step_size) {
    int i = idx / idx_slice_size;
    auto array_index = p_index[idx];
    p_value[idx] =
        p_array[i * array_slice_size + array_index];
  }
}

// UNDERSTAND: something like put_along_axis in numpy but if there is duplicate
// indices, scatter is done in += way.
template <typename T>
__global__ void GPUPutAlongD1(const int size,
                           const int batch_size,
                           const int num_take,
                           const int array_slice_size,
                           const int idx_slice_size,
                           T* p_array,
                           const int64_t* p_index,
                           const T* p_value) {
  const auto value_slice_size = idx_slice_size;
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  int step_size = blockDim.x * gridDim.x;

  for (; idx < size; idx += step_size) {
    int i = idx / idx_slice_size;
    auto array_index = p_index[idx];
    p_array[i * array_slice_size + array_index] = 
        p_value[idx];
  }
}


// UNDERSTAND: compute accidentdal hits from samples and minus corresponding
// logits by a float max, here 1e20
template <typename T>
__global__ void gpu_compute_remove_accidental_hits(const int size,
                           const int num_true,
                           const int idx_slice_size,
                           const int64_t* p_index,
                           T* p_value) {

  const auto value_slice_size = idx_slice_size;
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  int step_size = blockDim.x * gridDim.x;

  for (; idx < size; idx += step_size) {
    int i = idx / idx_slice_size;
    for (int j = 0; j < num_true; ++j) {
      const auto true_idx = i * idx_slice_size + j;
      if (p_index[true_idx] == p_index[idx]) {
        p_value[idx] -= 1e20;
        break;
      }
    }
  }
}

template <typename T>
class SampleLogitsCUDAKernel : 
      public framework::OpKernel<T> {
 public:
  using Tensor = framework::Tensor;
    void Print(Tensor & t, std::string name) const {
      VLOG(1) << "qxz print "<< name;
      VLOG(1) << name << "size = " << t.numel();
      size_t size = t.numel();
      T *d = t.data<T>();
    #ifdef PADDLE_WITH_CUDA
	std::vector<T> vec;
	platform::DeviceContextPool::Instance().Get(t.place())->Wait();
	if (platform::is_gpu_place(t.place())) {
	  vec.resize(size);
	  hipMemcpy(vec.data(), d, sizeof(T) * size, hipMemcpyDeviceToHost);
	  d = vec.data();
	}
    #endif
      VLOG(1) << name << " data_ptr = " << static_cast<void*>(d);
      for (size_t i = 0; i < size; i++) {
	   VLOG(1)<< d[i] << ",";
      }
    }

  void Compute(const framework::ExecutionContext& context) const override {
    // get necessary inputs
    const Tensor* logits = context.Input<Tensor>("Logits");
    const Tensor* label = context.Input<Tensor>("Label");
    VLOG(3) << "Enter SampleLogitsCUDAKernel";

    // get necessary outputs
    Tensor* samples = context.Output<Tensor>("Samples");
    Tensor* sampled_logits = context.Output<Tensor>("SampledLogits");
    Tensor* sampled_label = context.Output<Tensor>("SampledLabel");

    // shapes
    const auto batch_size = logits->dims()[0];
    const auto num_classes = logits->dims()[1];
    const auto label_dim = label->dims();
    const auto num_true = label_dim[1];
    const auto samples_dim = samples->dims();

    // attrs
    const auto num_samples = context.Attr<int>("num_samples");
    const bool use_custom_samples = context.Attr<bool>("use_custom_samples");
    const bool remove_accidental_hits =
        context.Attr<bool>("remove_accidental_hits");

    // device contexts
    auto& dev_ctx =
        context.template device_context<platform::CPUDeviceContext>();

    // UNDERSTAND: allocate memories for temporaries
    Tensor probabilities_tmp;
    Tensor* probabilities = &probabilities_tmp;
    sampled_logits->mutable_data<T>(samples_dim, context.GetPlace());
    auto sampled_label_data =
        sampled_label->mutable_data<int64_t>(label_dim, context.GetPlace());
    for (int i = 0; i < batch_size; ++i)
      for (int j = 0; j < num_true; ++j)
        sampled_label_data[i * num_true + j] = j;

    if (use_custom_samples) {
      const Tensor* custom_samples = context.Input<Tensor>("CustomSamples");
      const Tensor* custom_probabilities =
          context.Input<Tensor>("CustomProbabilities");
      samples->ShareDataWith(*custom_samples);
      probabilities->ShareDataWith(*custom_probabilities);
    } else {
      samples->mutable_data<int64_t>(context.GetPlace());
      probabilities->mutable_data<T>(samples_dim, context.GetPlace());
      // UNDERSTAND: sampling
      const auto seed = context.Attr<int>("seed");
      auto sampler_with_prob =
          math::GPUSampleWithProb<T>();
      Print(*samples, std::string("samples"));
      sampler_with_prob(context.cuda_device_context(), seed, num_classes,
          num_samples, label, samples, probabilities);
    }
    Print(*samples, std::string("samples"));
    Print(*probabilities, std::string("probabilities"));

    // UNDERSTAND: gather sampled logits and remove accidental hits if needed
    const auto num_take = samples->dims()[1];
    const auto array_dims = logits->dims();
    const auto idx_dims = samples->dims();

    const T* p_array = logits->data<T>();
    const int64_t* p_index = samples->data<int64_t>();
    T* p_value = sampled_logits->data<T>();
  
    // src slice size
    const auto array_slice_size = array_dims[1];
    // index slice size
    const auto idx_slice_size = idx_dims[1];

    int threads = 512;
    const size_t size = batch_size * num_take; 
    int grid = (size + threads - 1) / threads;

    GPUTakeAlongD1<T><<<grid, threads, 0, context.cuda_device_context().stream()>>>(
        size, batch_size, num_take, array_slice_size, idx_slice_size, p_array, p_index, p_value);
    if (remove_accidental_hits) {
      int threads = 512;
      const size_t size = batch_size * (num_true + num_samples); 
      int grid = (size + threads - 1) / threads;
      gpu_compute_remove_accidental_hits<T><<<grid, threads, 0, context.cuda_device_context().stream()>>>(size, num_true, idx_slice_size, p_index, p_value);
    }

    /* Debug
    const auto num_sampled_classes = samples_dim[1];
    std::cout << "Sampled Logits" << std::endl;
    const auto sampled_logits_data = sampled_logits->data<T>();
    for (int i = 0; i < sampled_logits->numel(); ++i) {
      std::cout << sampled_logits_data[i] << ", ";
      if ((i + 1) % num_sampled_classes == 0)
        std::cout << std::endl;
    }
    std::cout << std::endl;
    */
    /* Debug
    std::cout << "Samples" << std::endl;
    const auto samples_data = samples->data<int64_t>();
    for (int i = 0; i < samples->numel(); ++i) {
      std::cout << samples_data[i] << ", ";
      if ((i + 1) % num_sampled_classes == 0)
        std::cout << std::endl;
    }
    std::cout << std::endl;
    */
    /* Debug
    std::cout << "Probabilities" << std::endl;
    const auto probabilities_data = probabilities->data<T>();
    for (int i = 0; i < probabilities->numel(); ++i) {
      std::cout << probabilities_data[i] << ", ";
      if ((i + 1) % num_sampled_classes == 0)
        std::cout << std::endl;
    }
    std::cout << std::endl;
    */
    // subtracted sampled logits with logQ(y|x)
    auto probs = EigenMatrix<T>::From(*probabilities);
    auto smp_logits = EigenMatrix<T>::From(*sampled_logits);
    smp_logits.device(*dev_ctx.eigen_device()) =
        (smp_logits - probs.log().unaryExpr(TolerableValue<T>()))
            .unaryExpr(TolerableValue<T>());

  }
};

template <typename T>
class SampleLogitsGradCUDAKernel : public framework::OpKernel<T> {
 public:
  using Tensor = framework::Tensor;
  void Compute(const framework::ExecutionContext& context) const override {
    auto logits_grad = context.Output<Tensor>(framework::GradVarName("Logits"));
    const Tensor* samples = context.Input<Tensor>("Samples");
    const Tensor*  sampled_logits_grad = context.Input<Tensor>(framework::GradVarName("SampledLogits"));
    logits_grad->mutable_data<T>(context.GetPlace());

    auto& dev_ctx =
        context.template device_context<platform::CPUDeviceContext>();
    math::SetConstant<platform::CPUDeviceContext, T> set_zero;
    set_zero(dev_ctx, logits_grad, static_cast<T>(0));

    // UNDERSTAND: scatter it back to logit_grad
    const auto batch_size = samples->dims()[0];
    const auto num_put = samples->dims()[1];
    const auto array_dims = logits_grad->dims();
    const auto idx_dims = samples->dims();

    T* p_array = logits_grad->data<T>();
    const int64_t* p_index = samples->data<int64_t>();
    const T* p_value = sampled_logits_grad->data<T>();
  
    // src slice size
    const auto array_slice_size = array_dims[1];
    // index slice size
    const auto idx_slice_size = idx_dims[1];

    int threads = 512;
    const size_t size = batch_size * num_put; 
    int grid = (size + threads - 1) / threads;

    GPUPutAlongD1<T><<<grid, threads, 0, context.cuda_device_context().stream()>>>(size, batch_size, num_put, array_slice_size, idx_slice_size, p_array, p_index, p_value);
  }
};

}  // namespace operators
}  // namespace paddle
namespace ops = paddle::operators;

REGISTER_OP_CUDA_KERNEL(sample_logits,
                       ops::SampleLogitsCUDAKernel<float>,
                       ops::SampleLogitsCUDAKernel<double>);
REGISTER_OP_CUDA_KERNEL(sample_logits_grad,
                       ops::SampleLogitsGradCUDAKernel<float>,
                       ops::SampleLogitsGradCUDAKernel<double>);
