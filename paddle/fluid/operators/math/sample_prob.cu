#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#pragma once
#include <iostream>
#include <vector>
#include <thrust/random.h>
#include <thrust/sort.h>

#include "paddle/fluid/framework/ddim.h"
#include "paddle/fluid/framework/eigen.h"
#include "paddle/fluid/framework/tensor.h"
#include "paddle/fluid/framework/operator.h"
#include "paddle/fluid/operators/math/sampler.h"
#include "paddle/fluid/operators/math/sample_prob.h"
namespace paddle {
namespace operators {
namespace math {

using Tensor = framework::Tensor;

template <typename T>
__device__ T gpu_adjust_prob(const T prob, const int num_samples, const int num_tries) {
  if (num_samples == num_tries) {
    return prob * num_samples;
  } else {
    return -expm1(num_tries * log1p(-prob));
  }
}

class GPULogUniformSampler{
 public:
  __device__ int64_t Sample(float random, const int range, const float log_range) const;
  __device__ float Probability(int64_t value, const float log_range) const;
};

__device__ int64_t GPULogUniformSampler::Sample(float random, const int range, const float log_range) const {
  // Got Log Uniform distribution from uniform distribution by
  // inverse_transform_sampling method
  const int64_t value =
      static_cast<int64_t>(exp(random * log_range)) - 1;
  // Mathematically, value should be <= range_, but might not be due to some
  // floating point roundoff, so we mod by range_.
  return value % range;
}

__device__ float GPULogUniformSampler::Probability(int64_t value, const float log_range) const {
  // Given f(x) = 1/[(x+1) * log_range_]
  // The value's  probability  is integral of f(x) from value to (value + 1)
  return (log((value + 2.0) / (value + 1.0))) / log_range;
}


template<typename T>
__global__ void SamplingCondidate(const size_t n, const int seed, const int range, 
    const float log_range, const int num_true, const std::size_t num_samples, 
    const int64_t* label_data, int64_t* samples_data, T* probabilities_data) {
  thrust::minstd_rand rng;
  rng.seed(seed);
  thrust::uniform_real_distribution<float> dist(0, 1);
  const int num_sampled_classes = num_true + num_samples;

  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  int step_size = 0;

  for (; idx < n; idx += blockDim.x * gridDim.x) {
    if (step_size == 0) {
      rng.discard(idx);
      step_size = blockDim.x * gridDim.x;
    } else {
      rng.discard(step_size);
    }
    int col_cnt = idx % num_sampled_classes;
    GPULogUniformSampler sampler;
    if (col_cnt < num_true) {
      samples_data[idx] = label_data[(idx / num_sampled_classes) * num_true + col_cnt];
    } else {
      samples_data[idx] = sampler.Sample(dist(rng), range, log_range);
    }
    probabilities_data[idx] = sampler.Probability(samples_data[idx], range);
    probabilities_data[idx] = gpu_adjust_prob(
        probabilities_data[idx], num_samples, num_samples);
  } 
}

template<typename T>
__global__ void UniqSamplingCondidate(const size_t n, const int seed, const int range, 
    const float log_range, const int num_true, const std::size_t num_samples, 
    const int64_t* label_data, int64_t* samples_data, T* probabilities_data) {
  thrust::minstd_rand rng;
  rng.seed(seed);
  thrust::uniform_real_distribution<float> dist(0, 1);
  const int num_sampled_classes = num_true + num_samples;

  int row_idx = blockDim.x * blockIdx.x + threadIdx.x;
  int step_size = 0;
  int row_size = num_samples + num_true;
  GPULogUniformSampler sampler;
 
  // n == batch_size
  for (; row_idx < n; row_idx += blockDim.x * gridDim.x) {
    if (step_size == 0) {
      rng.discard(row_idx * num_samples);
      step_size = blockDim.x * gridDim.x * num_samples;
    } else {
      rng.discard(step_size);
    }
    // temp sets for unique sampling
    int num_tries = 0;
    int j = 0;
    while (j < num_true) {
      int idx = row_idx * row_size + j;
      samples_data[idx] = label_data[row_idx * num_true + j];
      probabilities_data[idx] = sampler.Probability(samples_data[idx], range);
      ++j;
    }

    while (j < num_samples + num_true) {
      ++num_tries;
      int idx = row_idx * row_size + j;
      auto v = sampler.Sample(dist(rng), range, log_range);
      bool find = false;
      for(int t = num_true;t < j ; t + 1) {
        if(samples_data[t] == v) {
            find = true; 
        }
      }
      if (find == false) {
        samples_data[idx] = v;
        probabilities_data[idx] = sampler.Probability(samples_data[idx], range);
        ++j;
      }
    }

    for (int k = 0;k < num_samples + num_true;k++) {
      int idx = row_idx * row_size + k;
      probabilities_data[idx] = gpu_adjust_prob(
	  probabilities_data[idx], num_samples, num_tries);
    }
  } 
}

template <typename T>
void GPUSampleWithProb<T>::operator()(const platform::CUDADeviceContext& context, const int seed, const int dict_size, const bool uniq,
                  const std::size_t num_samples, const Tensor* L, Tensor* S,
                  Tensor* P) {
    // UNDERSTAND: dimension issues
    const auto lbl_dim = L->dims();
    const int batch_size = lbl_dim[0];
    const int num_true = lbl_dim[1];
    const int num_sampled_classes = num_true + num_samples;
    framework::DDim ret_dim{batch_size, num_sampled_classes};

    // UNDERSTAND: raw data view
    const int64_t* label_data = L->data<int64_t>();
    int64_t* samples_data =
        S->mutable_data<int64_t>(ret_dim, context.GetPlace());
    T* probabilities_data = P->mutable_data<T>(ret_dim, context.GetPlace());
    if (uniq) {
      int threads = 128;
      const size_t size = batch_size; 
      int grid = (size + threads - 1) / threads;
      UniqSamplingCondidate<T><<<grid, threads, 0, context.stream()>>>(size, seed, dict_size, log(dict_size), num_true, num_samples, label_data, samples_data, probabilities_data); 
    } else {
      int threads = 512;
      const size_t size = batch_size * num_sampled_classes; 
      int grid = (batch_size * num_sampled_classes + threads - 1) / threads;
      SamplingCondidate<T><<<grid, threads, 0, context.stream()>>>(size, seed, dict_size, log(dict_size), num_true, num_samples, label_data, samples_data, probabilities_data); 
    }
};

template class GPUSampleWithProb<float>;
template class GPUSampleWithProb<double>;
}  // namespace math
}  // namespace operators
}  // namespace paddle
